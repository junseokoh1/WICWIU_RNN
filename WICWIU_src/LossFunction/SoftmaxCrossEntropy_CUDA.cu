
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "SoftmaxCrossEntropy.hpp"

// template class SoftmaxCrossEntropy<int>;
template class SoftmaxCrossEntropy<float>;
// template class SoftmaxCrossEntropy<double>;

//작은 값을 선택해주는 inline 함수
/*
inline float min(float x, float y)
{
  return x>y ? y : x ;
}
*/

__global__ void SoftmaxCrossEntropy_ForwardPropagate_kernel(int time, int batchsize, int colsize, float epsilon, float *result, float *label, float *softmaxresult) {
    int result_idx = 0;
    int start      = 0;
    int end        = 0;

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < batchsize; idx += blockDim.x * gridDim.x) {
        result_idx = time * batchsize + idx;
        start      = result_idx * colsize;
        end        = start + colsize;
        //printf("colsize : %d\n", colsize);
        for (int i = start; i < end; i++) {
            //result[result_idx] += -label[i] * log(MIN(softmaxresult[i], softmaxresult[i] + epsilon));
            result[result_idx] += -label[i] * log(MAX(softmaxresult[i], softmaxresult[i] + epsilon));



            // if(isnan(result[result_idx]) != 0){
            //     printf("nan인 경우 index : %d\n", result_idx);
            //     printf("%f \n", MAX(softmaxresult[i], softmaxresult[i] + epsilon));
            //     printf("%f \n\n", log(MAX(softmaxresult[i], softmaxresult[i] + epsilon)));
            //
            // }


            //result[result_idx] += -label[i] * log(softmaxresult[i]);
        //    printf("\n %d \n", i);
        }
    }
}

__global__ void print_kernel(int time, int batchsize) {
    int result_idx = 0;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < batchsize; idx += blockDim.x * gridDim.x) {
        result_idx = time * batchsize + idx;
        printf("\n idx = %d result_idx =%d \n", idx, result_idx);
    }
}

template<typename DTYPE> Tensor<DTYPE> *SoftmaxCrossEntropy<DTYPE>::ForwardPropagateOnGPU(int pTime) {
    Tensor<DTYPE> *input         = this->GetTensor();
    Tensor<DTYPE> *label         = this->GetLabel()->GetResult();
    Tensor<DTYPE> *softmaxresult = m_aSoftmaxResult;
    Tensor<DTYPE> *result        = this->GetResult();

    #ifdef __LOSS__
      std::cout<<"SoftmaxCrossEntropy Forward 호출 time = "<<pTime<<'\n';
      std::cout<<"softmaxcrossentropy 의 입력값 : "<<'\n'<<input<<'\n';
      std::cout<<"softmaxcrossentropy 의 라벨 값 : "<<'\n'<<label<<'\n';
    #endif

    int batchsize = input->GetBatchSize();
    int colsize   = input->GetColSize();

    float alpha = 1.f;
    float beta  = 0.f;

    cudnnTensorDescriptor_t pInputDesc   = input->GetDescriptor();
    cudnnTensorDescriptor_t pSoftMaxDesc = softmaxresult->GetDescriptor();

    DTYPE *pDevInput   = input->GetGPUData(pTime);
    DTYPE *pDevSoftMax = softmaxresult->GetGPUData(pTime);

  //std::cout<<"softmax 실행 전의 결과"<<'\n';
  //std::cout<<softmaxresult<<'\n';

    checkCUDNN(cudnnSoftmaxForward(this->GetCudnnHandle(), CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_INSTANCE,
                                   &alpha, pInputDesc, pDevInput,
                                   &beta, pSoftMaxDesc, pDevSoftMax));

     // std::cout<<"kernel함수 전의 결과"<<'\n';
     // std::cout<<softmaxresult<<'\n';

    int noBlock = 3, threadsPerBlock = 128;
    GetKernelParameters(batchsize, &noBlock, &threadsPerBlock);

    DTYPE *pDevLabel  = label->GetGPUData(pTime);
    DTYPE *pDevResult = result->GetGPUData(pTime);

//    std::cout<<"softmaxcrossentropy의 계산전 result 값"<<'\n';
//    std::cout<<result<<'\n';

    SoftmaxCrossEntropy_ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (0, batchsize, colsize, m_epsilon, pDevResult, pDevLabel, pDevSoftMax);
    //print_kernel<<<noBlock, threadsPerBlock>>>(pTime, batchsize);


    // std::cout<<"softmaxcrossentropy의 계산 결과"<<'\n';
    // std::cout<<result<<'\n';

    return result;
}

__global__ void SoftmaxCrossEntropy_BackPropagate_kernel(int time, int capacity, float *input_delta, float *label, float *softmaxresult) {
    int idx = 0;


    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < capacity; idx += blockDim.x * gridDim.x) {
        idx = time * capacity + idx;

        input_delta[idx] = softmaxresult[idx] - label[idx];
    }
}

template<typename DTYPE> Tensor<DTYPE> *SoftmaxCrossEntropy<DTYPE>::BackPropagateOnGPU(int pTime) {
    Tensor<DTYPE> *label         = this->GetLabel()->GetResult();
    Tensor<DTYPE> *softmaxresult = m_aSoftmaxResult;
    Tensor<DTYPE> *input_delta   = this->GetOperator()->GetDelta();

    int batchsize = input_delta->GetBatchSize();
    int colsize   = input_delta->GetColSize();
    int capacity  = batchsize * colsize;

    DTYPE *pDevSoftMax    = softmaxresult->GetGPUData(pTime);
    DTYPE *pDevLabel      = label->GetGPUData(pTime);
    DTYPE *pDevInputDelta = input_delta->GetGPUData(pTime);

    int noBlock = 3, threadsPerBlock = 128;
    GetKernelParameters(capacity, &noBlock, &threadsPerBlock);

    SoftmaxCrossEntropy_BackPropagate_kernel << < noBlock, threadsPerBlock >> > (0, capacity, pDevInputDelta, pDevLabel, pDevSoftMax);

    return NULL;
}

#endif  // ifdef __CUDNN__
