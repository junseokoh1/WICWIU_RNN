
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "PRelu.hpp"

// template class PRelu<int>;
template class PRelu<float>;
// template class PRelu<double>;

/*!
@class PRelu cuda
*/


__global__ void ForwardPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevInput[idx] > 0.f)
                pDevOutput[idx] = pDevInput[idx];
          else
                pDevOutput[idx] = pDevWeight[idx]* pDevInput[idx];
    }
}


template<typename DTYPE> int PRelu<DTYPE>::ForwardPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *input  = this->GetInput()[0]->GetResult();
        Tensor<DTYPE> *weight  = this->GetInput()[1]->GetResult();
        Tensor<DTYPE> *result = this->GetResult();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevInput  = input->GetGPUData(pTime);
        DTYPE *m_pDevWeight  = weight->GetGPUData(pTime);
        DTYPE *m_pDevOutput = result->GetGPUData(pTime);

        ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevInput, m_pDevWeight, m_pDevOutput,  m_parameterDim);

        return TRUE;
}


__global__ void BackPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, float *pDevDelta, float *pDevInputDelta, float *pDevWeightDelta, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevOutput[idx] > 0.f){
                pDevInputDelta[idx] += pDevDelta[idx];
                pDevWeightDelta[idx] += 0;
          }
          else{
                pDevInputDelta[idx] += pDevWeight[idx]* pDevDelta[idx];
                pDevWeightDelta[idx] += pDevInput[idx]* pDevDelta[idx];
          }
    }
}


template<typename DTYPE> int PRelu<DTYPE>::BackPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *input  = this->GetInput()[0]->GetResult();
        Tensor<DTYPE> *weight  = this->GetInput()[1]->GetResult();
        Tensor<DTYPE> *result = this->GetResult();
        Tensor<DTYPE> *this_delta  = this->GetGradient();
        Tensor<DTYPE> *input_delta = this->GetInput()[0]->GetDelta();
        Tensor<DTYPE> *weight_delta = this->GetInput()[1]->GetDelta();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevInput = input->GetGPUData(pTime);
        DTYPE *m_pDevWeight  = weight->GetGPUData(pTime);
        DTYPE *m_pDevOutput = result->GetGPUData(pTime);

        DTYPE *m_pDevDelta      = this_delta->GetGPUData(pTime);
        DTYPE *m_pDevInputDelta = input_delta->GetGPUData(pTime);
        DTYPE *m_pDevWeightDelta = weight_delta->GetGPUData(pTime);

        BackPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevInput, m_pDevWeight, m_pDevOutput, m_pDevDelta, m_pDevInputDelta, m_pDevWeightDelta, m_parameterDim);

        return TRUE;
}

#endif  // ifdef __CUDNN__
