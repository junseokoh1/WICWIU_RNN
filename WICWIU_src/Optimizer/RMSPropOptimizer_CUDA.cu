
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "RMSPropOptimizer.hpp"

// template class RMSPropOptimizer<int>;
template class RMSPropOptimizer<float>;
// template class RMSPropOptimizer<double>;

//////////////////////////////////////////////////////////////////////////////// for private method

/*!
@brief 파라미터 값들을 업데이트 하는 커널함수
@details UpdateParameterOnGPU 생성자에서 호출되어 실행
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevWeight 업데이트 할 파라미터의 GPU data.
@param pDevAccGradient 업데이트 할 파라미터의 gradient.
@param weightDim 업데이트 할 파라미터의 dimension.
@param signed_learning_rate Optimizer의 학습률
@param decay MeanSqaured와 gradiet 제곱 값의 가중치
@param epsilon 분모가 0이 되는 것을 방지
@param weightDecayRate 가중치 매개변수가 클 때 패널티를 부과하는 값
@param pMeanSquared 업데이트 할 pMeanSquared
@see int RMSPropOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pMeanSquared)
*/
__global__ void RMSPropUpdate_kernel(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float decay, float epsilon, float weightDecayRate, float *pMeanSquared) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
        float g = pDevAccGradient[idx];
        pMeanSquared[idx] = (decay * pMeanSquared[idx]) + ((1.F - decay) * (g * g)); //meansquared

        pDevWeight[idx]     += signed_learning_rate * weightDecayRate * pDevWeight[idx];
        pDevWeight[idx]     += signed_learning_rate / sqrt(pMeanSquared[idx] + epsilon) * g;
        pDevAccGradient[idx] = 0.F;
    }
}

/*!
@brief 파라미터 값들을 업데이트 하는 커널함수
@details UpdateParameterOnGPU 생성자에서 호출되어 실행
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevWeight 업데이트 할 파라미터의 GPU data.
@param pDevAccGradient 업데이트 할 파라미터의 gradient.
@param weightDim 업데이트 할 파라미터의 dimension.
@param signed_learning_rate Optimizer의 학습률
@param decay MeanSqaured, pMeanGrad와 gradiet 제곱 값의 가중치
@param epsilon 분모가 0이 되는 것을 방지
@param weightDecayRate 가중치 매개변수가 클 때 패널티를 부과하는 값
@param pMeanSquared 업데이트 할 pMeanSquared
@param pMeanGrad 업데이트 할 pMeanGrad
@see int RMSPropOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pMeanSquared)
*/
__global__ void RMSPropUpdate_kernelForCentered(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float decay, float epsilon, float weightDecayRate, float *pMeanSquared, float *pMeanGrad) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
        float g = pDevAccGradient[idx];
        pMeanGrad[idx] = (decay * pMeanGrad[idx]) + ((1.f - decay) * g); //meangrad
        pMeanSquared[idx] = (decay * pMeanSquared[idx]) + ((1.F - decay) * (g * g)); //meansquared

        pDevWeight[idx]     += signed_learning_rate * weightDecayRate * pDevWeight[idx];
        pDevWeight[idx]     += signed_learning_rate / sqrt((pMeanSquared[idx] - (pMeanGrad[idx] * pMeanGrad[idx])) + epsilon) * g;

        pDevAccGradient[idx] = 0.F;
    }
}

/*!
@brief RMSPropOptimizer UpdateParameterOnGPU centered false 생성자.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread 갯수
@details m_parameterDim는 업데이트 할 파라미터의 dimension
@details m_pDevData, m_pDevGrad, m_pDevGradientSquared는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data 생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details RMSPropUpdate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU데이터를 다음과 같은 형식으로 호출.
@see __global__ void RMSPropUpdate_kernel(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float decay, float epsilon, float weightDecayRate, float *pMeanSquared)
@param *pParameter 업데이트 할 Tensor를 가지고 있는 Operator포인터
@param pMeanSquared 업데이트 할 pMeanSquared 변수
@return 성공 시 TRUE
*/
template<typename DTYPE> int RMSPropOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pMeanSquared) {
    int noBlock = 3, threadsPerBlock = 128;

    int m_parameterDim = pParameter->GetResult()->GetCapacity();

    GetKernelParameters(m_parameterDim, &noBlock, &threadsPerBlock);

    float signed_learning_rate = this->GetOptimizeDirection() * this->GetLearningRate();
    float weightDecayRate = this->GetWeightDecayRate();

    Tensor<DTYPE> *trainable_data = pParameter->GetResult();
    Tensor<DTYPE> *gradient       = pParameter->GetGradient();

    // std::cout<<"RMSProp updataParameter  "<<pParameter->GetName()<<'\n';
    // std::cout<<gradient->GetShape()<<'\n';
    // std::cout<<gradient<<'\n';

    DTYPE *m_pDevData          = trainable_data->GetGPUData();
    DTYPE *m_pDevGrad          = gradient->GetGPUData();
    DTYPE *m_pDevMeanSquared   = pMeanSquared->GetGPUData();

    RMSPropUpdate_kernel << < noBlock, threadsPerBlock >> > (m_pDevData, m_pDevGrad, m_parameterDim, signed_learning_rate, m_decay, m_epsilon, weightDecayRate, m_pDevMeanSquared);

    return TRUE;
}

/*!
@brief RMSPropOptimizer UpdateParameterOnGPU centered true 생성자.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread 갯수
@details m_parameterDim는 업데이트 할 파라미터의 dimension
@details m_pDevData, m_pDevGrad, m_pDevGradientSquared는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data 생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details RMSPropUpdate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU데이터를 다음과 같은 형식으로 호출.
@see __global__ void RMSPropUpdate_kernelForCentered(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float decay, float epsilon, float weightDecayRate, float *pMeanSquared, float *pMeanGrad)
@param pMeanSquared 업데이트 할 pMeanSquared 변수
@param pMeanGrad 업데이트 할 pMeanGrad 변수
@return 성공 시 TRUE
*/
template<typename DTYPE> int RMSPropOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pMeanSquared, Tensor<DTYPE> *pMeanGrad) {
    int noBlock = 3, threadsPerBlock = 128;

    int m_parameterDim = pParameter->GetResult()->GetCapacity();

    GetKernelParameters(m_parameterDim, &noBlock, &threadsPerBlock);

    float signed_learning_rate = this->GetOptimizeDirection() * this->GetLearningRate();
    float weightDecayRate = this->GetWeightDecayRate();

    Tensor<DTYPE> *trainable_data = pParameter->GetResult();
    Tensor<DTYPE> *gradient       = pParameter->GetGradient();

    DTYPE *m_pDevData          = trainable_data->GetGPUData();
    DTYPE *m_pDevGrad          = gradient->GetGPUData();
    DTYPE *m_pDevMeanSquared   = pMeanSquared->GetGPUData();
    DTYPE *m_pDevMeanGrad      = pMeanGrad->GetGPUData();

    RMSPropUpdate_kernelForCentered << < noBlock, threadsPerBlock >> > (m_pDevData, m_pDevGrad, m_parameterDim, signed_learning_rate, m_decay, m_epsilon, weightDecayRate, m_pDevMeanSquared, m_pDevMeanGrad);

    return TRUE;
}

#endif  // ifdef __CUDNN__
